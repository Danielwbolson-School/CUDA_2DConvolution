#include "hip/hip_runtime.h"
#ifndef _2DCONVOLUTION_KERNEL_H_
#define _2DCONVOLUTION_KERNEL_H_

#include <stdio.h>
#include "2Dconvolution.h"

const int32_t KERNEL_WIDTH = 5;
const int32_t numHalo = KERNEL_WIDTH / 2;
const int32_t tileWidth = 32;
const int32_t sharedTileWidth = tileWidth + KERNEL_WIDTH - 1;

// Constant tells the GPU to aggressively cache this variable. Cache coherency isn't an issue
// since these values won't ever change.
__constant__ float Kernel[KERNEL_SIZE][KERNEL_SIZE];

__host__ void ConstantInitialization(float* elements, int32_t size)
{
    hipMemcpyToSymbol(HIP_SYMBOL(Kernel), elements, size);
}

// Matrix multiplication kernel thread specification
__global__ void ConvolutionKernel(Matrix N, Matrix P)
{
    int32_t blockX = blockIdx.x; int32_t blockY = blockIdx.y;
    int32_t blockDimX = blockDim.x; int32_t blockDimY = blockDim.y;
    int32_t threadX = threadIdx.x; int32_t threadY = threadIdx.y;

    int32_t row = blockY * blockDimX + threadY;
    int32_t column = blockX * blockDimY + threadX;

    __shared__ float N_s[sharedTileWidth][sharedTileWidth];

    // Store in shared memory.

    // Setting up our shared memory is a bit funky in 2D.
    // Our threads only match up with the middle matrix, but we need to store all values
    // that are marked with an X.
    // +--+--+--+--+--+  +--+--+--+--+--+  +--+--+--+--+--+
    // |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--+--+--------+  +--------------+  +--------+--+--+
    
    // +--+--+--------+  +--------------+  +--------+--+--+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--+--+--------+  +--------------+  +--------+--+--+
    
    // +--+--+--------+  +--------------+  +--------+--+--+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |XX|XX|  |XX|XX|XX|XX|XX|  |XX|XX|  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
    // +--------------+  +--------------+  +--------------+
    // |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |  |
    // +--+--+--+--+--+  +--+--+--+--+--+  +--+--+--+--+--+
    
    // An if-check for all 8 surrounding grids. Quite a bit of divergence here, however the
    // memory access savings can potentially be huge, dependent on ratio of image size to kernel size.
    
    // Is it possible that our tile is smaller than the half width of our kernel? I am assuming that
    // tile size is enough to cover the kernel with my 3x3 grid operation below.

    // Only update values that change.

    // Up and Left
    int32_t blockThreadRow = (blockY - 1) * blockDimY + threadY;
    int32_t blockThreadCol = (blockX - 1) * blockDimX + threadX;
    int32_t sharedThreadRow = threadY - (blockDimY - numHalo);
    int32_t sharedThreadCol = threadX - (blockDimX - numHalo);
    int32_t rowOffset = 0; 
    int32_t colOffset = 0;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Up
    //blockThreadRow = (blockY - 1) * blockDimY + threadY;
    blockThreadCol = (blockX) * blockDimX + threadX;
    //sharedThreadRow = threadY - (blockDimY - numHalo);
    sharedThreadCol = threadX;
    //rowOffset = 0; 
    colOffset = numHalo;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Up and Right
    //blockThreadRow = (blockY - 1) * blockDimY + threadY;
    blockThreadCol = (blockX + 1) * blockDimX + threadX;
    //sharedThreadRow = threadY - (blockDimY - numHalo);
   // sharedThreadCol = threadX;
    //rowOffset = 0; 
    colOffset = numHalo + blockDimX;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol < numHalo) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Left
    blockThreadRow = (blockY) * blockDimY + threadY;
    blockThreadCol = (blockX - 1) * blockDimX + threadX;
    sharedThreadRow = threadY;
    sharedThreadCol = threadX - (blockDimX - numHalo);
    rowOffset = numHalo; 
    colOffset = 0;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Center
    //blockThreadRow = (blockY) * blockDimY + threadY;
    blockThreadCol = (blockX) * blockDimX + threadX;
    //sharedThreadRow = threadY;
    sharedThreadCol = threadX;
    //rowOffset = numHalo; 
    colOffset = numHalo;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Right
    //blockThreadRow = (blockY) * blockDimY + threadY;
    blockThreadCol = (blockX + 1) * blockDimX + threadX;
    //sharedThreadRow = threadY;
    //sharedThreadCol = threadX;
    //rowOffset = numHalo; 
    colOffset = numHalo + blockDimX;

    if (
        (sharedThreadRow >= 0) && 
        (sharedThreadCol < numHalo) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Down and Left
    blockThreadRow = (blockY + 1) * blockDimY + threadY;
    blockThreadCol = (blockX - 1) * blockDimX + threadX;
    //sharedThreadRow = threadY;
    sharedThreadCol = threadX - (blockDimX - numHalo);
    rowOffset = numHalo + blockDimY; 
    colOffset = 0;

    if (
        (sharedThreadRow < numHalo) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Down
    //blockThreadRow = (blockY + 1) * blockDimY + threadY;
    blockThreadCol = (blockX) * blockDimX + threadX;
    //sharedThreadRow = threadY;
    sharedThreadCol = threadX;
    //rowOffset = numHalo + blockDimY; 
    colOffset = numHalo;

    if (
        (sharedThreadRow < numHalo) && 
        (sharedThreadCol >= 0) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }

    // Down and Right
    //blockThreadRow = (blockY + 1) * blockDimY + threadY;
    blockThreadCol = (blockX + 1) * blockDimX + threadX;
    //sharedThreadRow = threadY;
    //sharedThreadCol = threadX;
    //rowOffset = numHalo + blockDimY; 
    colOffset = numHalo + blockDimX;

    if (
        (sharedThreadRow < numHalo) && 
        (sharedThreadCol < numHalo) && 
        (sharedThreadRow + rowOffset < sharedTileWidth) && 
        (sharedThreadCol + colOffset < sharedTileWidth)
    )
    {
        N_s[sharedThreadRow + rowOffset][sharedThreadCol + colOffset] = 
            ((blockThreadRow < 0) || (blockThreadRow >= N.height) || (blockThreadCol < 0) || (blockThreadCol >= N.width)) ?
                0 : N.elements[blockThreadRow * N.width + blockThreadCol];
    }
    __syncthreads();
    
    if (row < P.height && column < P.width)
    {
        // Actual convolution math.
        float pValue = 0;
        
        for (int32_t i = 0; i < KERNEL_WIDTH; i++)
        {
            for (int32_t j = 0; j < KERNEL_WIDTH; j++)
            {
                pValue += Kernel[i][j] * N_s[threadY + i][threadX + j];
            }
        }

        P.elements[row * P.width + column] = pValue;

    }
}

#endif // #ifndef _2DCONVOLUTION_KERNEL_H_
